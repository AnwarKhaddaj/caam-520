
#include <hip/hip_runtime.h>
const int n_rows = 1280;
const int n_cols = 320;
const int N_cols = n_cols / 32;
const int N = 64;

__global__ void matrix_scale_wrapped(float* A, float* B, float* D, const int n_rows, const int n_cols, const int N);

__device__ int cartesian_2_flat(int i, int j, int stride) { //helper function
    return i*stride+j;
}

void launch_gpu_matrix_scale(float* A_cpu, float* B_cpu, float* D_cpu) {
    // Pointers for memory on the GPU
    float* A_gpu;
    float* D_gpu;
    float* B_gpu;

    // Allocate memory on the GPU
    hipMalloc(&A_gpu,n_rows*n_cols*sizeof(float));
    hipMalloc(&D_gpu,n_cols*sizeof(float));
    hipMalloc(&B_gpu,n_rows*n_cols*sizeof(float));

    // Copy memory to the GPU
    hipMemcpy(A_gpu,A_cpu,n_rows*n_cols*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(D_gpu,D_cpu,n_cols*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(B_gpu,B_cpu,n_rows*n_cols*sizeof(float),hipMemcpyHostToDevice);

    // Compute how much shared memory is needed
    int shared_mem = 32*sizeof(float);

    // Compute the size of the CUDA grid (i.e. the number of blocks in each dim)
    int threads_per_block = 32;
    dim3 number_of_blocks(n_rows/N,N_cols);

    // Invoke the CUDA kernel
    matrix_scale_wrapped<<<number_of_blocks,threads_per_block,shared_mem>>>(A_gpu, B_gpu, D_gpu, n_rows, n_cols, N);

    // Copy memory back to the CPU
    hipMemcpy(A_cpu,A_gpu,n_rows*n_cols*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(B_cpu,B_gpu,n_rows*n_cols*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(D_cpu,D_gpu,n_cols*sizeof(float),hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(D_gpu);
}

__global__ void matrix_scale_wrapped(float* A, float* B, float* D, const int n_rows, const int n_cols, const int N) {
    
    int i = threadIdx.x;
    int I = blockIdx.x;
    int J = blockIdx.y;
    
    int Dy = blockDim.y;

    __shared__ float shared_mem[32]; //shared memory
    shared_mem[i] = D[i+J*Dy]; //copying memory from a chunk of D to shared memory

    int el_ind;  //index of the element in A that the thread needs to access
    for (int warp_jump = 0; warp_jump < N; warp_jump++) {
        el_ind = cartesian_2_flat(I*N+warp_jump,J*Dy+i,n_cols); //jumping by one warp as threads access vertical elements
        B[el_ind] = A[el_ind]*shared_mem[warp_jump];
    }
}

int main() {
    // Pointers for the matrices/arrays
    float* A;
    float* B;
    float* D;

    // Allocate the arrays
    A = new float[n_rows*n_cols];
    B = new float[n_rows*n_cols];
    D = new float[n_cols];

    for (int i = 0; i < n_rows; i++) {
        for (int j = 0; j < n_cols; j++) {
            A[j + i*n_cols] = 1;
        }
    }

    for(int j = 0; j < n_cols; j++)
        D[j] = j;

    launch_gpu_matrix_scale(A, B, D);

    // Check B
    int correct = 0;
    for (int i = 0; i < n_rows; i++) {
        for (int j = 0; j < n_cols; j++) {
            if (B[j + i *n_cols] != j)
                correct = -1;
        }
    }

    delete[] A;
    delete[] B;
    delete[] D;
    
    return correct;
}